#include "hip/hip_runtime.h"
#include "stdio.h"
#include <cuda/std/atomic>

__global__
void plus(unsigned char *ptc){
  *ptc+=1;
}

__global__
void less(unsigned char *ptc){
  *ptc-=1;
}

__global__
void morethan(unsigned char *ptc){
  ptc++;
}

__global__
void lessthan(unsigned char *ptc){
  ptc--;
}

void dot(unsigned char *ptc){
  putchar(*ptc);
}

void comma(unsigned char *ptc){
  *ptc = getchar();
}

int main(){
  unsigned char tape[30000] = {0};
  unsigned char *ptr = tape;
  size_t ptr_size = sizeof(char);
  hipMallocManaged(&ptr, ptr_size);
  char input[10000]; // Input of 10k chars max
  hipFree(ptr);
  return 0;
}
